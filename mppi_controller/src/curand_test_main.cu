#include <hip/hip_runtime.h>

#include <hiprand/hiprand.h>
#include <vector>
#include <iostream>


int main(int argc, char** argv) {

    hipStream_t optimization_stride;// = 1;//getRosParam<int>("optimization_stride", node_);
    hipStreamCreate(&optimization_stride);

    hiprandGenerator_t gen_;
    hiprandCreateGenerator(&gen_, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen_, 1234ULL);

    hiprandSetStream(gen_,optimization_stride);

    float* du_d_;

    hipMalloc((void **)& du_d_, 100*sizeof(float));

    hiprandGenerateNormal(gen_, du_d_, 100, 0.0, 1.0);

    float* du_ = (float*)malloc(100*sizeof(float));

    hipMemcpy(du_, du_d_, 100*sizeof(float), hipMemcpyDeviceToHost);
    
    for (int i = 0; i < 100; i++) {
        std::cout << *(du_+i) << std::endl;
    }

    std::cout << std::endl;

}